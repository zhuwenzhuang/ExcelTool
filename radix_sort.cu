#include "hip/hip_runtime.h"
#include "utils.h"

/* Red Eye Removal
   ===============
   
   For this assignment we are implementing red eye removal.  This is
   accomplished by first creating a score for every pixel that tells us how
   likely it is to be a red eye pixel.  We have already done this for you - you
   are receiving the scores and need to sort them in ascending order so that we
   know which pixels to alter to remove the red eye.

   Note: ascending order == smallest to largest

   Each score is associated with a position, when you sort the scores, you must
   also move the positions accordingly.

   Implementing Parallel Radix Sort with CUDA
   ==========================================

   The basic idea is to construct a histogram on each pass of how many of each
   "digit" there are.   Then we scan this histogram so that we know where to put
   the output of each digit.  For example, the first 1 must come after all the
   0s so we have to know how many 0s there are to be able to start moving 1s
   into the correct position.

   1) Histogram of the number of occurrences of each digit
                    [0 2 3 5 4 6 1]=>[4 3] 针对pass的 0 1
   2) Exclusive Prefix Sum of Histogram
                    [4 7]
   3) Determine relative offset of each digit
        For example [0 0 1 1 0 0 1]
                ->  [0 1 0 1 2 3 2]
   4) Combine the results of steps 2 & 3 to determine the final
      output location for each element and move it there

   LSB Radix sort is an out-of-place sort and you will need to ping-pong values
   between the input and output buffers we have provided.  Make sure the final
   sorted results end up in the output buffer!  Hint: You may need to do a copy
   at the end.

 */
// scan in 1024 size section.
__global__
void scanSections(unsigned int* const d_inputVals,
                  unsigned int* const d_scan,
                  unsigned int bit,
                  size_t numElems)
{
    unsigned int offset = blockDim.x * blockIdx.x;
    int idx = threadIdx.x + offset;
    if(idx >= numElems){
        return ;
    }
    // map 0 => 1, 1 in scan mean little
    d_scan[idx] = (d_inputVals[idx]&bit) == 0?1:0;
    __syncthreads();
    
    // Inclusive Hillis-Steele scan
    unsigned int val = 0;
    for(int i=1; i < blockDim.x; i*=2){
        val = idx >= (i + offset) ? d_scan[idx - i] + d_scan[idx] : d_scan[idx];
        __syncthreads();
        d_scan[idx] = val;
        __syncthreads();
    }
}

__global__
void scanHighestValues(unsigned int* d_scan, 
                       unsigned int* d_scanHighest,
                       unsigned int blockSize)
{
    int idx = threadIdx.x;
    // |   |   |    |
    d_scanHighest[idx] = idx == 0 ? 0 : d_scan[idx*blockSize -1];
    __syncthreads();
    unsigned int val = 0;
    for(int i=1;i<blockDim.x;i*=2){
        val = idx >= i ? d_scanHighest[idx] + d_scanHighest[idx - i] : d_scanHighest[idx];
        __syncthreads();
        d_scanHighest[idx] = val; 
        __syncthreads();
    }
}
//6666666 分层累加
__global__ 
void scanMerge(unsigned int* d_scan, unsigned int* d_scanHighest, unsigned int blockSize,size_t numElems)
{
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if(idx >= numElems){
        return;
    }
    
    unsigned int highestIdx = idx / blockSize;
    d_scan[idx] += d_scanHighest[highestIdx];
}

__global__
void move(unsigned int* const d_inputVals,
          unsigned int* const d_inputPos,
          unsigned int* const d_outputVals,
          unsigned int* const d_outputPos,
          unsigned int* const d_scan,
          const size_t numElems,          
          unsigned int bit)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    if(x >= numElems){
        return ;
    }
    unsigned int startPos = d_scan[numElems - 1];
    unsigned int index = 0;
    if((d_inputVals[x] & bit) == 0){
        index = x == 0 ? 0 : d_scan[x - 1];
    }else{
        index = startPos + x - (x == 0 ? 0:d_scan[x - 1]);
    }
    d_outputVals[index] = d_inputVals[x];
    d_outputPos[index] = d_inputPos[x];
}

void your_sort(unsigned int* const d_inputVals,
               unsigned int* const d_inputPos,
               unsigned int* const d_outputVals,
               unsigned int* const d_outputPos,
               const size_t numElems)
{
   const dim3 gridSize(numElems/1024 + 1);
   const dim3 blockSize(1024);
   
   unsigned int* d_scan;
   unsigned int* d_scanHighest;
   checkCudaErrors(hipMalloc(&d_scan, sizeof(unsigned int) * numElems));
   checkCudaErrors(hipMalloc(&d_scanHighest, sizeof(unsigned int) * gridSize.x));
   
   for(unsigned int i = 0;i<8*sizeof(unsigned int);i++){
        unsigned int bit = 1 << i;
        // Since the maximum number of threads per block is 1024, an
        // array of hundreds of thousands values can not be scanned at
        // once. Thus scan is done in three parts:
        // 1. Create scan array for each section according to block size (e.g. 0-1023, 1024-2047 etc.)
        // 2. Create another scan array from the last values of each block (highest values)
        // 3. Add the highest scan values to the original scan array to form one continous scan array
        
        scanSections<<<gridSize, blockSize>>>(d_inputVals, d_scan, bit, numElems);
        scanHighestValues<<<1, gridSize>>>(d_scan,d_scanHighest,blockSize.x);
        scanMerge<<<gridSize, blockSize>>>(d_scan, d_scanHighest, blockSize.x, numElems);
   
        move<<<gridSize, blockSize>>>(d_inputVals, d_inputPos, d_outputVals, d_outputPos,
                                      d_scan,numElems, bit);
        checkCudaErrors(hipMemcpy(d_inputVals, d_outputVals, numElems * sizeof(unsigned int), hipMemcpyDeviceToDevice));
        checkCudaErrors(hipMemcpy(d_inputPos, d_outputPos, numElems * sizeof(unsigned int), hipMemcpyDeviceToDevice));
        
   }
   hipFree(d_scan);
   hipFree(d_scanHighest);
}
